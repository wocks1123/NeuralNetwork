#include "hip/hip_runtime.h"
//test.cu
#include "CudaTest/CudaTest.hpp"
#include <stdio.h>

#include <hip/hip_runtime_api.h>
#include <>

__global__ void sum_kernel(int a, int b, int *c)
{
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
        printf("tid : %d\n", tid);
        c[tid] = a + b;
}

int CudaTest::sum_gpu(int a, int b, int *c)
{
        int *f;
        hipMalloc((void**)&f, sizeof(int)* 1);
        hipMemcpy(f, c, sizeof(int)* 1, hipMemcpyHostToDevice);

        sum_kernel <<<1, 1>>>(a, b, f);
        hipMemcpy(c, f, sizeof(int)* 1, hipMemcpyDeviceToHost);
        hipFree(f);

        return true;
}

